#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 277: GPU Programmming 2020
/* Author and Instructer: Cheolhong An
/* Copyright 2020
/* University of California, San Diego
/*************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernel_relu(float* A, float* O, int M, int N);

void cu_relu(float* A, float* O, int M, int N)
{
	float* d_a, * d_o;

	dim3 blk;
	blk.x = 16; blk.y = 16; blk.z = 1;

	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;
	grid.z = 1;

	int size = sizeof(unsigned int) * M * N;

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_o, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);

	kernel_relu << < grid, blk >> > (d_a, d_o, M, N);

	hipMemcpy(O, d_o, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_o);
}

__global__ void kernel_relu(float* A, float* O, int M, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < M * N) {
		O[index] = fmaxf(A[index], 0);
	}
}
